#include "hip/hip_runtime.h"
#include <thrust/distance.h>
#include <thrust/execution_policy.h>
#include <thrust/for_each.h>
#include <thrust/tabulate.h>

#include <cstdint>

#include <common.cuh>
#include <hiprand.h>

class generator_t
{
private:
  generator_t();

public:

  static generator_t &instance();
  ~generator_t();

  template <typename T>
  void operator()(seed_t seed,
                  thrust::device_vector<T> &data,
                  T min = std::numeric_limits<T>::min(),
                  T max = std::numeric_limits<T>::max());

  float* distribution();
  hiprandGenerator_t &gen() { return m_gen; }

  float* prepare_random_generator(
      seed_t seed,
      std::size_t num_items);

private:
  hiprandGenerator_t m_gen;
  thrust::device_vector<float> m_distribution;
};

generator_t& generator_t::instance()
{
  static generator_t generator;
  return generator;
}

template <typename T>
struct random_to_item_t
{
  float m_min;
  float m_max;

  __host__ __device__ random_to_item_t(T min, T max)
      : m_min(static_cast<float>(min))
      , m_max(static_cast<float>(max))
  {}

  __device__ T operator()(float random_value)
  {
    return static_cast<T>((m_max - m_min) * random_value + m_min);
  }
};

generator_t::generator_t()
{
  hiprandCreateGenerator(&m_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
}

generator_t::~generator_t()
{
  hiprandDestroyGenerator(m_gen);
}

float* generator_t::distribution()
{
  return thrust::raw_pointer_cast(m_distribution.data());
}

float *generator_t::prepare_random_generator(seed_t seed, 
                                             std::size_t num_items)
{
  hiprandSetPseudoRandomGeneratorSeed(m_gen, seed.get());

  m_distribution.resize(num_items);
  hiprandGenerateUniform(m_gen,
                        this->distribution(),
                        num_items);

  return this->distribution();
}

template <class T>
void generator_t::operator()(seed_t seed,
                             thrust::device_vector<T> &data,
                             T min,
                             T max)
{
  prepare_random_generator(seed, data.size());

  thrust::transform(m_distribution.begin(),
                    m_distribution.end(),
                    data.begin(),
                    random_to_item_t<T>(min, max));
}

template <typename T>
void gen(seed_t seed, 
         thrust::device_vector<T> &data,
         T min,
         T max)
{
  generator_t::instance()(seed, data, min, max);
}

#define INSTANTIATE_RND(TYPE) \
template \
void gen<TYPE>( \
    seed_t, \
    thrust::device_vector<TYPE> &data, \
    TYPE min, \
    TYPE max)

#define INSTANTIATE(TYPE) \
  INSTANTIATE_RND(TYPE); 

INSTANTIATE(std::uint8_t);
INSTANTIATE(std::uint16_t);
INSTANTIATE(std::uint32_t);
INSTANTIATE(std::uint64_t);
INSTANTIATE(__uint128_t);

INSTANTIATE(std::int8_t);
INSTANTIATE(std::int16_t);
INSTANTIATE(std::int32_t);
INSTANTIATE(std::int64_t);
INSTANTIATE(__int128_t);

