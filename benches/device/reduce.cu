#include <cub/device/device_reduce.cuh>

#include <common.cuh>

// %PARAM% TUNE_ITEMS_PER_THREAD ipt 7:8:9:10:11:12:13:14:15:16:17:18:19:20:21:22:23:24
// %PARAM% TUNE_THREADS_PER_BLOCK tpb 128:160:192:224:256:288:320:352:384:416:448:480:512:544:576:608:640:672:704:736:768:800:832:864:896:928:960:992:1024
// %PARAM% TUNE_ITEMS_PER_VEC_LOAD ipv 1:2:4

#if !TUNE_BASE
template <typename AccumT, typename OffsetT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int threads_per_block  = TUNE_THREADS_PER_BLOCK;
    static constexpr int items_per_thread   = TUNE_ITEMS_PER_THREAD;
    static constexpr int items_per_vec_load = TUNE_ITEMS_PER_VEC_LOAD;

    using ReducePolicy = hipcub::AgentReducePolicy<threads_per_block,
                                                items_per_thread,
                                                AccumT,
                                                items_per_vec_load,
                                                hipcub::BLOCK_REDUCE_WARP_REDUCTIONS,
                                                hipcub::LOAD_DEFAULT>;

    // SingleTilePolicy
    using SingleTilePolicy = ReducePolicy;

    // SegmentedReducePolicy
    using SegmentedReducePolicy = ReducePolicy;
  };

  using MaxPolicy = policy_t;
};
#endif

template <typename T, typename OffsetT>
void reduce(nvbench::state &state, nvbench::type_list<T, OffsetT>)
{
  using accum_t     = T;
  using input_it_t  = const T *;
  using output_it_t = T *;
  using offset_t    = typename hipcub::detail::ChooseOffsetT<OffsetT>::Type;
  using output_t    = T;
  using init_t      = T;
  using op_t        = hipcub::Sum;
#if !TUNE_BASE
  using policy_t   = policy_hub_t<accum_t, offset_t>;
  using dispatch_t =
    hipcub::DispatchReduce<input_it_t, output_it_t, offset_t, op_t, init_t, accum_t, policy_t>;
#else
  using dispatch_t =
    hipcub::DispatchReduce<input_it_t, output_it_t, offset_t, op_t, init_t, accum_t>;
#endif

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> in(elements);
  thrust::device_vector<T> out(1);

  gen(seed_t{}, in);

  input_it_t d_in   = thrust::raw_pointer_cast(in.data());
  output_it_t d_out = thrust::raw_pointer_cast(out.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(1);

  // Allocate temporary storage:
  std::size_t temp_size;
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_in,
                       d_out,
                       static_cast<offset_t>(elements),
                       op_t{},
                       init_t{},
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         d_in,
                         d_out,
                         static_cast<offset_t>(elements),
                         op_t{},
                         init_t{},
                         launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(reduce, NVBENCH_TYPE_AXES(all_value_types, offset_types))
  .set_name("hipcub::DeviceReduce::Reduce")
  .set_type_axes_names({"T", "OffsetT"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

