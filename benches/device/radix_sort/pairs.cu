#include <cub/device/device_radix_sort.cuh>

#include <thrust/device_vector.h>

#include <string>

#include <common.cuh>
#include <type_traits>

// %PARAM% TUNE_RADIX_BITS bits 6:7:8
// %PARAM% TUNE_ITEMS_PER_THREAD ipt 9:10:11:12:13:14:15:16:17:18:19:20:21:22:23:24:25
// %PARAM% TUNE_THREADS_PER_BLOCK tpb 96:128:160:192:224:256:288:320:352:384:416:448:480:512:544:576:608:640:672:704:736:768:800:832:864:896:928:960:992:1024

constexpr bool is_descending   = false;
constexpr bool is_overwrite_ok = false;

#if !TUNE_BASE
template <typename KeyT, typename ValueT, typename OffsetT>
struct policy_hub_t
{
  constexpr static bool KEYS_ONLY = std::is_same<ValueT, hipcub::NullType>::value;

  using DominantT = hipcub::detail::conditional_t<(sizeof(ValueT) > sizeof(KeyT)), ValueT, KeyT>;

  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int ONESWEEP_RADIX_BITS = TUNE_RADIX_BITS;
    static constexpr bool ONESWEEP           = true;
    static constexpr bool OFFSET_64BIT       = sizeof(OffsetT) == 8;

    // Onesweep policy
    using OnesweepPolicy =
      hipcub::AgentRadixSortOnesweepPolicy<TUNE_THREADS_PER_BLOCK,
                                        TUNE_ITEMS_PER_THREAD,
                                        DominantT,
                                        1,
                                        hipcub::RADIX_RANK_MATCH_EARLY_COUNTS_ANY,
                                        hipcub::BLOCK_SCAN_RAKING_MEMOIZE,
                                        hipcub::RADIX_SORT_STORE_DIRECT,
                                        ONESWEEP_RADIX_BITS>;

    // These kernels are launched once, no point in tuning at the moment
    using HistogramPolicy = hipcub::AgentRadixSortHistogramPolicy<128, 16, 1, KeyT, ONESWEEP_RADIX_BITS>;
    using ExclusiveSumPolicy = hipcub::AgentRadixSortExclusiveSumPolicy<256, ONESWEEP_RADIX_BITS>;
    using ScanPolicy = hipcub::AgentScanPolicy<512,
                                            23,
                                            OffsetT,
                                            hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                            hipcub::LOAD_DEFAULT,
                                            hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                                            hipcub::BLOCK_SCAN_RAKING_MEMOIZE>;

    // No point in tuning
    static constexpr int SINGLE_TILE_RADIX_BITS = (sizeof(KeyT) > 1) ? 6 : 5;

    // No point in tuning single-tile policy
    using SingleTilePolicy = hipcub::AgentRadixSortDownsweepPolicy<256,
                                                                19,
                                                                DominantT,
                                                                hipcub::BLOCK_LOAD_DIRECT,
                                                                hipcub::LOAD_LDG,
                                                                hipcub::RADIX_RANK_MEMOIZE,
                                                                hipcub::BLOCK_SCAN_WARP_SCANS,
                                                                SINGLE_TILE_RADIX_BITS>;
  };

  using MaxPolicy = policy_t;
};

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_onesweep_temp_storage_size()
{
  using portion_offset  = int;
  using onesweep_policy = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::OnesweepPolicy;
  using agent_radix_sort_onesweep_t = cub::
    AgentRadixSortOnesweep<onesweep_policy, is_descending, KeyT, ValueT, OffsetT, portion_offset>;

  return sizeof(typename agent_radix_sort_onesweep_t::TempStorage);
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_temp_storage_size()
{
  using policy_t = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t;

  static_assert(policy_t::ONESWEEP);
  return max_onesweep_temp_storage_size<KeyT, ValueT, OffsetT>();
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr bool fits_in_default_shared_memory()
{
  return max_temp_storage_size<KeyT, ValueT, OffsetT>() < 48 * 1024;
}
#else
template <typename, typename, typename>
constexpr bool fits_in_default_shared_memory()
{
  return true;
}
#endif

template <typename T, typename OffsetT>
void radix_sort_values(std::integral_constant<bool, true>,
                       nvbench::state &state,
                       nvbench::type_list<T, OffsetT>)
{
  using offset_t = typename hipcub::detail::ChooseOffsetT<OffsetT>::Type;

  using key_t   = T;
  using value_t = T;
#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t, value_t, offset_t>;
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t, policy_t>;
#else
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t>;
#endif

  const int begin_bit = 0;
  const int end_bit   = sizeof(key_t) * 8;

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> keys_buffer_1(elements);
  thrust::device_vector<T> keys_buffer_2(elements);
  thrust::device_vector<T> values_buffer_1(elements);
  thrust::device_vector<T> values_buffer_2(elements);

  key_t *d_keys_buffer_1     = thrust::raw_pointer_cast(keys_buffer_1.data());
  key_t *d_keys_buffer_2     = thrust::raw_pointer_cast(keys_buffer_2.data());
  value_t *d_values_buffer_1 = thrust::raw_pointer_cast(values_buffer_1.data());
  value_t *d_values_buffer_2 = thrust::raw_pointer_cast(values_buffer_2.data());

  gen(seed_t{}, keys_buffer_1);
  gen(seed_t{}, values_buffer_1);

  hipcub::DoubleBuffer<key_t> d_keys(d_keys_buffer_1, d_keys_buffer_2);
  hipcub::DoubleBuffer<value_t> d_values(d_values_buffer_1, d_values_buffer_2);

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements * 2, "Size");
  state.add_global_memory_writes<T>(elements * 2);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_keys,
                       d_values,
                       static_cast<offset_t>(elements),
                       begin_bit,
                       end_bit,
                       is_overwrite_ok,
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    hipcub::DoubleBuffer<key_t> keys     = d_keys;
    hipcub::DoubleBuffer<value_t> values = d_values;

    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         keys,
                         values,
                         static_cast<offset_t>(elements),
                         begin_bit,
                         end_bit,
                         is_overwrite_ok,
                         launch.get_stream());
  });
}

template <typename T, typename OffsetT>
void radix_sort_values(std::integral_constant<bool, false>,
                       nvbench::state &,
                       nvbench::type_list<T, OffsetT>)
{
  (void)is_descending;
  (void)is_overwrite_ok;
}

template <typename T, typename OffsetT>
void radix_sort_values(nvbench::state &state, nvbench::type_list<T, OffsetT> tl)
{
  using offset_t = typename hipcub::detail::ChooseOffsetT<OffsetT>::Type;

  radix_sort_values(std::integral_constant<bool, fits_in_default_shared_memory<T, T, offset_t>()>{},
                    state,
                    tl);
}

NVBENCH_BENCH_TYPES(radix_sort_values, NVBENCH_TYPE_AXES(all_value_types, offset_types))
  .set_name("hipcub::DeviceRadixSort::SortPairs")
  .set_type_axes_names({"T", "OffsetT"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

