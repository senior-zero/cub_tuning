#include "hip/hip_runtime.h"
#include <cub/device/device_radix_sort.cuh>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <string>

#include <nvbench/nvbench.cuh>
#include <type_traits>

// %PARAM% TUNE_RADIX_BITS bits 5:6:7:8

using value_t  = hipcub::NullType;
using offset_t = std::int32_t;

constexpr bool is_descending   = false;
constexpr bool is_overwrite_ok = true;

#if !TUNE_BASE
template <typename KeyT, typename ValueT, typename OffsetT>
struct policy_hub_t
{
  constexpr static bool KEYS_ONLY = std::is_same<ValueT, hipcub::NullType>::value;

  using DominantT = hipcub::detail::conditional_t<(sizeof(ValueT) > sizeof(KeyT)), ValueT, KeyT>;

  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    enum
    {
      PRIMARY_RADIX_BITS     = TUNE_RADIX_BITS,
      SINGLE_TILE_RADIX_BITS = (sizeof(KeyT) > 1) ? 6 : 5, // No point in tuning
      SEGMENTED_RADIX_BITS   = TUNE_RADIX_BITS,
      ONESWEEP               = sizeof(KeyT) >= sizeof(uint32_t),
      ONESWEEP_RADIX_BITS    = TUNE_RADIX_BITS,
      OFFSET_64BIT           = sizeof(OffsetT) == 8,
    };

    // Histogram policy
    typedef hipcub::AgentRadixSortHistogramPolicy<128, 16, 1, KeyT, ONESWEEP_RADIX_BITS>
      HistogramPolicy;

    // Exclusive sum policy
    typedef hipcub::AgentRadixSortExclusiveSumPolicy<256, ONESWEEP_RADIX_BITS> ExclusiveSumPolicy;

    // Onesweep policy
    typedef hipcub::AgentRadixSortOnesweepPolicy<384,
                                              OFFSET_64BIT && sizeof(KeyT) == 4 && !KEYS_ONLY ? 17
                                                                                              : 21,
                                              DominantT,
                                              1,
                                              hipcub::RADIX_RANK_MATCH_EARLY_COUNTS_ANY,
                                              hipcub::BLOCK_SCAN_RAKING_MEMOIZE,
                                              hipcub::RADIX_SORT_STORE_DIRECT,
                                              ONESWEEP_RADIX_BITS>
      OnesweepPolicy;

    // ScanPolicy
    typedef hipcub::AgentScanPolicy<512,
                                 23,
                                 OffsetT,
                                 hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                 hipcub::LOAD_DEFAULT,
                                 hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                                 hipcub::BLOCK_SCAN_RAKING_MEMOIZE>
      ScanPolicy;

    // Downsweep policies
    typedef hipcub::AgentRadixSortDownsweepPolicy<512,
                                               23,
                                               DominantT,
                                               hipcub::BLOCK_LOAD_TRANSPOSE,
                                               hipcub::LOAD_DEFAULT,
                                               hipcub::RADIX_RANK_MATCH,
                                               hipcub::BLOCK_SCAN_WARP_SCANS,
                                               PRIMARY_RADIX_BITS>
      DownsweepPolicy;
    typedef hipcub::AgentRadixSortDownsweepPolicy<(sizeof(KeyT) > 1) ? 256 : 128,
                                               47,
                                               DominantT,
                                               hipcub::BLOCK_LOAD_TRANSPOSE,
                                               hipcub::LOAD_DEFAULT,
                                               hipcub::RADIX_RANK_MEMOIZE,
                                               hipcub::BLOCK_SCAN_WARP_SCANS,
                                               PRIMARY_RADIX_BITS - 1>
      AltDownsweepPolicy;

    // Upsweep policies
    typedef hipcub::AgentRadixSortUpsweepPolicy<256, 23, DominantT, hipcub::LOAD_DEFAULT, PRIMARY_RADIX_BITS>
      UpsweepPolicy;
    typedef cub::
      AgentRadixSortUpsweepPolicy<256, 47, DominantT, hipcub::LOAD_DEFAULT, PRIMARY_RADIX_BITS - 1>
        AltUpsweepPolicy;

    // Single-tile policy
    typedef hipcub::AgentRadixSortDownsweepPolicy<256,
                                               19,
                                               DominantT,
                                               hipcub::BLOCK_LOAD_DIRECT,
                                               hipcub::LOAD_LDG,
                                               hipcub::RADIX_RANK_MEMOIZE,
                                               hipcub::BLOCK_SCAN_WARP_SCANS,
                                               SINGLE_TILE_RADIX_BITS>
      SingleTilePolicy;
  };

  using MaxPolicy = policy_t;
};

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_upsweep_temp_storage_size()
{
  using upsweep_policy = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::UpsweepPolicy;
  using alt_upsweep_policy =
    typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::AltUpsweepPolicy;

  using agent_radix_sort_upsweep_t = hipcub::AgentRadixSortUpsweep<upsweep_policy, KeyT, OffsetT>;
  using alt_agent_radix_sort_upsweep_t =
    hipcub::AgentRadixSortUpsweep<alt_upsweep_policy, KeyT, OffsetT>;

  return hipcub::max(sizeof(typename agent_radix_sort_upsweep_t::TempStorage),
                  sizeof(typename alt_agent_radix_sort_upsweep_t::TempStorage));
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_downsweep_temp_storage_size()
{
  using downsweep_policy = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::DownsweepPolicy;
  using alt_downsweep_policy =
    typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::AltDownsweepPolicy;

  using agent_radix_sort_downsweep_t =
    hipcub::AgentRadixSortDownsweep<downsweep_policy, is_descending, KeyT, ValueT, OffsetT>;
  using alt_agent_radix_sort_downsweep_t =
    hipcub::AgentRadixSortDownsweep<alt_downsweep_policy, is_descending, KeyT, ValueT, OffsetT>;

  return hipcub::max(sizeof(typename agent_radix_sort_downsweep_t::TempStorage),
                  sizeof(typename alt_agent_radix_sort_downsweep_t::TempStorage));
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_onesweep_temp_storage_size()
{
  using portion_offset  = int;
  using onesweep_policy = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t::OnesweepPolicy;
  using agent_radix_sort_onesweep_t = cub::
    AgentRadixSortOnesweep<onesweep_policy, is_descending, KeyT, ValueT, OffsetT, portion_offset>;

  return sizeof(typename agent_radix_sort_onesweep_t::TempStorage);
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr std::size_t max_temp_storage_size()
{
  using policy_t = typename policy_hub_t<KeyT, ValueT, OffsetT>::policy_t;

  return policy_t::ONESWEEP ? max_onesweep_temp_storage_size<KeyT, ValueT, OffsetT>()
                            : hipcub::max(max_upsweep_temp_storage_size<KeyT, ValueT, OffsetT>(),
                                       max_downsweep_temp_storage_size<KeyT, ValueT, OffsetT>());
}

template <typename KeyT, typename ValueT, typename OffsetT>
constexpr bool fits_in_default_shared_memory()
{
  return max_temp_storage_size<KeyT, ValueT, OffsetT>() < 48 * 1024;
}
#else
template <typename, typename, typename>
constexpr bool fits_in_default_shared_memory()
{
  return true;
}
#endif

template <typename T>
void radix_sort_keys(std::integral_constant<bool, true>,
                     nvbench::state &state,
                     nvbench::type_list<T>)
{
  using key_t = T;
#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t, value_t, offset_t>;
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t, policy_t>;
#else
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t>;
#endif

  const int begin_bit = 0;
  const int end_bit   = sizeof(key_t) * 8;

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> buffer_1(elements);
  thrust::device_vector<T> buffer_2(elements);
  thrust::sequence(buffer_1.begin(), buffer_1.end());
  thrust::sequence(buffer_2.begin(), buffer_2.end());

  key_t *d_buffer_1 = thrust::raw_pointer_cast(buffer_1.data());
  key_t *d_buffer_2 = thrust::raw_pointer_cast(buffer_2.data());

  hipcub::DoubleBuffer<key_t> d_keys(d_buffer_1, d_buffer_2);
  hipcub::DoubleBuffer<value_t> d_values;

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_keys,
                       d_values,
                       static_cast<offset_t>(elements),
                       begin_bit,
                       end_bit,
                       is_overwrite_ok,
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         d_keys,
                         d_values,
                         static_cast<offset_t>(elements),
                         begin_bit,
                         end_bit,
                         is_overwrite_ok,
                         launch.get_stream());
  });
}

template <typename T>
void radix_sort_keys(std::integral_constant<bool, false>, nvbench::state &, nvbench::type_list<T>)
{
  (void)is_descending;
  (void)is_overwrite_ok;
}

template <typename T>
void radix_sort_keys(nvbench::state &state, nvbench::type_list<T> tl)
{
  radix_sort_keys(
    std::integral_constant<bool, fits_in_default_shared_memory<T, value_t, offset_t>()>{},
    state,
    tl);
}

using all_value_types =
  nvbench::type_list<nvbench::int8_t, nvbench::int16_t, nvbench::int32_t, nvbench::int64_t, __int128_t>;

NVBENCH_BENCH_TYPES(radix_sort_keys, NVBENCH_TYPE_AXES(all_value_types))
  .set_name("hipcub::DeviceRadixSort::SortKeys")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

