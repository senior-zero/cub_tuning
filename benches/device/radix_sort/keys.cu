#include <cub/device/device_radix_sort.cuh>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <string>

#include <nvbench/nvbench.cuh>
#include <type_traits>

// %PARAM% TUNE_RADIX_BITS bits 5

#if !TUNE_BASE
template <typename KeyT, typename ValueT, typename OffsetT>
struct policy_hub_t
{
  constexpr static bool KEYS_ONLY = std::is_same<ValueT, hipcub::NullType>::value;

  using DominantT = hipcub::detail::conditional_t<(sizeof(ValueT) > sizeof(KeyT)), ValueT, KeyT>;

  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    enum
    {
      PRIMARY_RADIX_BITS     = (sizeof(KeyT) > 1) ? 7 : 5,
      SINGLE_TILE_RADIX_BITS = (sizeof(KeyT) > 1) ? 6 : 5,
      SEGMENTED_RADIX_BITS   = (sizeof(KeyT) > 1) ? 6 : 5,
      ONESWEEP               = sizeof(KeyT) >= sizeof(uint32_t),
      ONESWEEP_RADIX_BITS    = 8,
      OFFSET_64BIT           = sizeof(OffsetT) == 8,
    };

    // Histogram policy
    typedef hipcub::AgentRadixSortHistogramPolicy<128, 16, 1, KeyT, ONESWEEP_RADIX_BITS>
      HistogramPolicy;

    // Exclusive sum policy
    typedef hipcub::AgentRadixSortExclusiveSumPolicy<256, ONESWEEP_RADIX_BITS> ExclusiveSumPolicy;

    // Onesweep policy
    typedef hipcub::AgentRadixSortOnesweepPolicy<384,
                                              OFFSET_64BIT && sizeof(KeyT) == 4 && !KEYS_ONLY ? 17
                                                                                              : 21,
                                              DominantT,
                                              1,
                                              hipcub::RADIX_RANK_MATCH_EARLY_COUNTS_ANY,
                                              hipcub::BLOCK_SCAN_RAKING_MEMOIZE,
                                              hipcub::RADIX_SORT_STORE_DIRECT,
                                              ONESWEEP_RADIX_BITS>
      OnesweepPolicy;

    // ScanPolicy
    typedef hipcub::AgentScanPolicy<512,
                                 23,
                                 OffsetT,
                                 hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                 hipcub::LOAD_DEFAULT,
                                 hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                                 hipcub::BLOCK_SCAN_RAKING_MEMOIZE>
      ScanPolicy;

    // Downsweep policies
    typedef hipcub::AgentRadixSortDownsweepPolicy<512,
                                               23,
                                               DominantT,
                                               hipcub::BLOCK_LOAD_TRANSPOSE,
                                               hipcub::LOAD_DEFAULT,
                                               hipcub::RADIX_RANK_MATCH,
                                               hipcub::BLOCK_SCAN_WARP_SCANS,
                                               PRIMARY_RADIX_BITS>
      DownsweepPolicy;
    typedef hipcub::AgentRadixSortDownsweepPolicy<(sizeof(KeyT) > 1) ? 256 : 128,
                                               47,
                                               DominantT,
                                               hipcub::BLOCK_LOAD_TRANSPOSE,
                                               hipcub::LOAD_DEFAULT,
                                               hipcub::RADIX_RANK_MEMOIZE,
                                               hipcub::BLOCK_SCAN_WARP_SCANS,
                                               PRIMARY_RADIX_BITS - 1>
      AltDownsweepPolicy;

    // Upsweep policies
    typedef hipcub::AgentRadixSortUpsweepPolicy<256, 23, DominantT, hipcub::LOAD_DEFAULT, PRIMARY_RADIX_BITS>
      UpsweepPolicy;
    typedef cub::
      AgentRadixSortUpsweepPolicy<256, 47, DominantT, hipcub::LOAD_DEFAULT, PRIMARY_RADIX_BITS - 1>
        AltUpsweepPolicy;

    // Single-tile policy
    typedef hipcub::AgentRadixSortDownsweepPolicy<256,
                                               19,
                                               DominantT,
                                               hipcub::BLOCK_LOAD_DIRECT,
                                               hipcub::LOAD_LDG,
                                               hipcub::RADIX_RANK_MEMOIZE,
                                               hipcub::BLOCK_SCAN_WARP_SCANS,
                                               SINGLE_TILE_RADIX_BITS>
      SingleTilePolicy;
  };

  using MaxPolicy = policy_t;
};
#endif

template <typename T>
void radix_sort_keys(nvbench::state &state, nvbench::type_list<T>)
{
  constexpr bool is_descending   = false;
  constexpr bool is_overwrite_ok = true;

  using key_t    = T;
  using value_t  = hipcub::NullType;
  using offset_t = std::int32_t;
#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t, value_t, offset_t>;
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t, policy_t>;
#else
  using dispatch_t = hipcub::DispatchRadixSort<is_descending, key_t, value_t, offset_t>;
#endif

  const int begin_bit = 0;
  const int end_bit   = sizeof(key_t) * 8;

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> buffer_1(elements);
  thrust::device_vector<T> buffer_2(elements);
  thrust::sequence(buffer_1.begin(), buffer_1.end());
  thrust::sequence(buffer_2.begin(), buffer_2.end());

  key_t *d_buffer_1 = thrust::raw_pointer_cast(buffer_1.data());
  key_t *d_buffer_2 = thrust::raw_pointer_cast(buffer_2.data());

  hipcub::DoubleBuffer<key_t> d_keys(d_buffer_1, d_buffer_2);
  hipcub::DoubleBuffer<value_t> d_values;

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_keys,
                       d_values,
                       static_cast<offset_t>(elements),
                       begin_bit,
                       end_bit,
                       is_overwrite_ok,
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         d_keys,
                         d_values,
                         static_cast<offset_t>(elements),
                         begin_bit,
                         end_bit,
                         is_overwrite_ok,
                         launch.get_stream());
  });
}

using all_value_types =
  nvbench::type_list<nvbench::int8_t, nvbench::int16_t, nvbench::int32_t, nvbench::int64_t>; // TODO
                                                                                             // __int128

NVBENCH_BENCH_TYPES(radix_sort_keys, NVBENCH_TYPE_AXES(all_value_types))
  .set_name("hipcub::DeviceRadixSort::SortKeys")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

