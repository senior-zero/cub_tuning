#include <cub/device/device_scan.cuh>

#include <common.cuh>

// %PARAM% TUNE_ITEMS_PER_THREAD ipt 7:8:9:10:11:12:13:14:15:16:17:18:19:20:21:22:23:24
// %PARAM% TUNE_THREADS_PER_BLOCK tpb 128:160:192:224:256:288:320:352:384:416:448:480:512:544:576:608:640:672:704:736:768:800:832:864:896:928:960:992:1024

#if !TUNE_BASE
template <typename AccumT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using ScanPolicyT = hipcub::AgentScanPolicy<TUNE_THREADS_PER_BLOCK,
                                             TUNE_ITEMS_PER_THREAD,
                                             AccumT,
                                             hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                             hipcub::LOAD_DEFAULT,
                                             hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                                             hipcub::BLOCK_SCAN_WARP_SCANS>;
  };

  using MaxPolicy = policy_t;
};
#endif

template <typename T>
static void basic(nvbench::state &state, nvbench::type_list<T>)
{
  using accum_t     = T;
  using input_it_t  = const T *;
  using output_it_t = T *;
  using offset_t    = std::int32_t;
  using output_t    = T;
  using init_t      = hipcub::detail::InputValue<T>;
  using op_t        = hipcub::Sum;

#if !TUNE_BASE
  using policy_t    = policy_hub_t<accum_t>;
  using dispatch_t =
    hipcub::DispatchScan<input_it_t, output_it_t, op_t, init_t, offset_t, accum_t, policy_t>;
#else
  using dispatch_t =
    hipcub::DispatchScan<input_it_t, output_it_t, op_t, init_t, offset_t, accum_t>;
#endif

  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<T> input(elements);
  thrust::device_vector<T> output(elements);

  gen(seed_t{}, input);

  T *d_input  = thrust::raw_pointer_cast(input.data());
  T *d_output = thrust::raw_pointer_cast(output.data());

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  size_t tmp_size;
  dispatch_t::Dispatch(nullptr,
                       tmp_size,
                       d_input,
                       d_output,
                       op_t{},
                       init_t{T{}},
                       static_cast<int>(input.size()),
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> tmp(tmp_size);
  nvbench::uint8_t *d_tmp = thrust::raw_pointer_cast(tmp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(thrust::raw_pointer_cast(tmp.data()),
                         tmp_size,
                         d_input,
                         d_output,
                         op_t{},
                         init_t{T{}},
                         static_cast<int>(input.size()),
                         launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(all_value_types))
  .set_name("hipcub::DeviceScan::ExclusiveSum")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 2));

