#include <cub/device/device_scan.cuh>
#include <type_traits>

#include <common.cuh>

// %PARAM% TUNE_ITEMS_PER_THREAD ipt 7:8:9:10:11:12:13:14:15:16:17:18:19:20:21:22:23:24
// %PARAM% TUNE_THREADS_PER_BLOCK tpb 128:160:192:224:256:288:320:352:384:416:448:480:512:544:576:608:640:672:704:736:768:800:832:864:896:928:960:992:1024

#if !TUNE_BASE
template <typename AccumT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using ScanPolicyT = hipcub::AgentScanPolicy<TUNE_THREADS_PER_BLOCK,
                                             TUNE_ITEMS_PER_THREAD,
                                             AccumT,
                                             hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                             hipcub::LOAD_DEFAULT,
                                             hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                                             hipcub::BLOCK_SCAN_WARP_SCANS>;
  };

  using MaxPolicy = policy_t;
};

template <typename T, typename OffsetT>
constexpr std::size_t max_temp_storage_size()
{
  using accum_t     = T;
  using input_it_t  = const T *;
  using output_it_t = T *;
  using offset_t    = OffsetT;
  using output_t    = T;
  using init_t      = hipcub::detail::InputValue<T>;
  using op_t        = hipcub::Sum;
  using policy_t    = typename policy_hub_t<accum_t>::policy_t;
  using real_init_t = typename init_t::value_type;

  using agent_scan_t =
    hipcub::AgentScan<typename policy_t::ScanPolicyT, 
                   input_it_t, 
                   output_it_t, 
                   op_t, 
                   real_init_t, 
                   offset_t, 
                   accum_t>;

  return sizeof(typename agent_scan_t::TempStorage);
}

template <typename T, typename OffsetT>
constexpr bool fits_in_default_shared_memory()
{
  return max_temp_storage_size<T, OffsetT>() < 48 * 1024;
}
#else
template <typename T, typename OffsetT>
constexpr bool fits_in_default_shared_memory()
{
  return true;
}
#endif

template <typename T, typename OffsetT>
static void basic(std::integral_constant<bool, true>,
                  nvbench::state &state,
                  nvbench::type_list<T, OffsetT>)
{
  using accum_t     = T;
  using input_it_t  = const T *;
  using output_it_t = T *;
  using offset_t    = OffsetT;
  using output_t    = T;
  using init_t      = hipcub::detail::InputValue<T>;
  using op_t        = hipcub::Sum;

#if !TUNE_BASE
  using policy_t = policy_hub_t<accum_t>;
  using dispatch_t =
    hipcub::DispatchScan<input_it_t, output_it_t, op_t, init_t, offset_t, accum_t, policy_t>;
#else
  using dispatch_t = hipcub::DispatchScan<input_it_t, output_it_t, op_t, init_t, offset_t, accum_t>;
#endif

  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<T> input(elements);
  thrust::device_vector<T> output(elements);

  gen(seed_t{}, input);

  T *d_input  = thrust::raw_pointer_cast(input.data());
  T *d_output = thrust::raw_pointer_cast(output.data());

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  size_t tmp_size;
  dispatch_t::Dispatch(nullptr,
                       tmp_size,
                       d_input,
                       d_output,
                       op_t{},
                       init_t{T{}},
                       static_cast<int>(input.size()),
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> tmp(tmp_size);
  nvbench::uint8_t *d_tmp = thrust::raw_pointer_cast(tmp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(thrust::raw_pointer_cast(tmp.data()),
                         tmp_size,
                         d_input,
                         d_output,
                         op_t{},
                         init_t{T{}},
                         static_cast<int>(input.size()),
                         launch.get_stream());
  });
}

template <typename T, typename OffsetT>
static void basic(std::integral_constant<bool, false>,
                  nvbench::state &,
                  nvbench::type_list<T, OffsetT>)
{
  // TODO Support
}

template <typename T, typename OffsetT>
static void basic(nvbench::state &state, nvbench::type_list<T, OffsetT> tl)
{
  basic(std::integral_constant<bool, (sizeof(OffsetT) == 4) && fits_in_default_shared_memory<T, OffsetT>()>{},
        state,
        tl);
}

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(all_value_types, offset_types))
  .set_name("hipcub::DeviceScan::ExclusiveSum")
  .set_type_axes_names({"T", "OffsetT"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 2));

