#include <cub/device/device_scan.cuh>

#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <nvbench/nvbench.cuh>

// %PARAM% TUNE_BLOCK_THREADS bt 128:256
// %PARAM% TUNE_ITEMS_PER_THREAD ipt 12:15

#if !TUNE_BASE
template <typename AccumT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using ScanPolicyT = hipcub::AgentScanPolicy<TUNE_BLOCK_THREADS,
                                             TUNE_ITEMS_PER_THREAD,
                                             AccumT,
                                             hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                             hipcub::LOAD_DEFAULT,
                                             hipcub::BLOCK_STORE_WARP_TRANSPOSE,
                                             hipcub::BLOCK_SCAN_WARP_SCANS>;
  };

  using MaxPolicy = policy_t;
};
#endif

template <typename T>
static void basic(nvbench::state &state, nvbench::type_list<T>)
{
  using accum_t     = T;
  using input_it_t  = const T *;
  using output_it_t = T *;
  using offset_t    = std::int32_t;
  using output_t    = T;
  using init_t      = hipcub::detail::InputValue<T>;
  using op_t        = hipcub::Sum;

#if !TUNE_BASE
  using policy_t    = policy_hub_t<accum_t>;
  using dispatch_t =
    hipcub::DispatchScan<input_it_t, output_it_t, op_t, init_t, offset_t, accum_t, policy_t>;
#else
  using dispatch_t =
    hipcub::DispatchScan<input_it_t, output_it_t, op_t, init_t, offset_t, accum_t>;
#endif

  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));

  thrust::device_vector<T> input(elements);
  thrust::device_vector<T> output(elements);
  thrust::sequence(input.begin(), input.end());

  T *d_input  = thrust::raw_pointer_cast(input.data());
  T *d_output = thrust::raw_pointer_cast(output.data());

  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  size_t tmp_size;
  dispatch_t::Dispatch(nullptr,
                       tmp_size,
                       d_input,
                       d_output,
                       op_t{},
                       init_t{T{}},
                       static_cast<int>(input.size()),
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> tmp(tmp_size);
  nvbench::uint8_t *d_tmp = thrust::raw_pointer_cast(tmp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(thrust::raw_pointer_cast(tmp.data()),
                         tmp_size,
                         d_input,
                         d_output,
                         op_t{},
                         init_t{T{}},
                         static_cast<int>(input.size()),
                         launch.get_stream());
  });
}

using types =
  nvbench::type_list<nvbench::int8_t, nvbench::int16_t, nvbench::int32_t, nvbench::int64_t, __int128>;

NVBENCH_BENCH_TYPES(basic, NVBENCH_TYPE_AXES(types))
  .set_name("hipcub::DeviceScan::ExclusiveSum")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 2));

