#include <cub/device/device_merge_sort.cuh>

#include <thrust/device_vector.h>
#include <thrust/sequence.h>

#include <string>
#include <type_traits>

#include <common.cuh>

// %PARAM% TUNE_BLOCK_THREADS bt 128:256:512
// %PARAM% TUNE_ITEMS_PER_THREAD ipt 16:17:18:19:20

using value_t  = hipcub::NullType;
using offset_t = std::int32_t;

#if !TUNE_BASE
template <typename KeyT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using MergeSortPolicy =
      hipcub::AgentMergeSortPolicy<TUNE_BLOCK_THREADS,
                                hipcub::Nominal4BItemsToItems<KeyT>(TUNE_ITEMS_PER_THREAD),
                                hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                hipcub::LOAD_DEFAULT,
                                hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
  };

  using MaxPolicy = policy_t;
};
#endif

struct less_t
{
  template <typename DataType>
  __device__ bool operator()(const DataType &lhs, const DataType &rhs)
  {
    return lhs < rhs;
  }
};

template <typename T>
void merge_sort_keys(nvbench::state &state, nvbench::type_list<T>)
{
  using key_t            = T;
  using value_t          = hipcub::NullType;
  using key_input_it_t   = key_t *;
  using value_input_it_t = value_t *;
  using key_it_t         = key_t *;
  using value_it_t       = value_t *;
  using offset_t         = int;
  using compare_op_t     = less_t;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t>;
  using dispatch_t = hipcub::DispatchMergeSort<key_input_it_t,
                                            value_input_it_t,
                                            key_it_t,
                                            value_it_t,
                                            offset_t,
                                            compare_op_t,
                                            policy_t>;
#else
  using dispatch_t = 
    hipcub::DispatchMergeSort<key_input_it_t, value_input_it_t, key_it_t, value_it_t, offset_t, compare_op_t>;
#endif

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> buffer_1(elements);
  thrust::device_vector<T> buffer_2(elements);
  thrust::sequence(buffer_1.rbegin(), buffer_1.rend());
  thrust::sequence(buffer_2.rbegin(), buffer_2.rend());

  key_t *d_buffer_1 = thrust::raw_pointer_cast(buffer_1.data());
  key_t *d_buffer_2 = thrust::raw_pointer_cast(buffer_2.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_buffer_1,
                       nullptr,
                       d_buffer_2,
                       nullptr,
                       static_cast<offset_t>(elements),
                       compare_op_t{},
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         d_buffer_1,
                         nullptr,
                         d_buffer_2,
                         nullptr,
                         static_cast<offset_t>(elements),
                         compare_op_t{},
                         launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(merge_sort_keys, NVBENCH_TYPE_AXES(all_value_types))
  .set_name("hipcub::DeviceMergeSort::SortKeys")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

