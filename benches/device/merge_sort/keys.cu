#include <cub/device/device_merge_sort.cuh>

#include <common.cuh>

// %PARAM% TUNE_ITEMS_PER_THREAD ipt 7:8:9:10:11:12:13:14:15:16:17:18:19:20:21:22:23:24
// %PARAM% TUNE_THREADS_PER_BLOCK tpb 64:128:256:512

using value_t  = hipcub::NullType;

#if !TUNE_BASE
template <typename KeyT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    using MergeSortPolicy =
      hipcub::AgentMergeSortPolicy<TUNE_THREADS_PER_BLOCK,
                                hipcub::Nominal4BItemsToItems<KeyT>(TUNE_ITEMS_PER_THREAD),
                                hipcub::BLOCK_LOAD_WARP_TRANSPOSE,
                                hipcub::LOAD_DEFAULT,
                                hipcub::BLOCK_STORE_WARP_TRANSPOSE>;
  };

  using MaxPolicy = policy_t;
};
#endif

struct less_t
{
  template <typename DataType>
  __device__ bool operator()(const DataType &lhs, const DataType &rhs)
  {
    return lhs < rhs;
  }
};

template <typename T, typename OffsetT>
void merge_sort_keys(nvbench::state &state, nvbench::type_list<T, OffsetT>)
{
  using key_t            = T;
  using value_t          = hipcub::NullType;
  using key_input_it_t   = key_t *;
  using value_input_it_t = value_t *;
  using key_it_t         = key_t *;
  using value_it_t       = value_t *;
  using offset_t         = OffsetT;
  using compare_op_t     = less_t;

#if !TUNE_BASE
  using policy_t   = policy_hub_t<key_t>;
  using dispatch_t = hipcub::DispatchMergeSort<key_input_it_t,
                                            value_input_it_t,
                                            key_it_t,
                                            value_it_t,
                                            offset_t,
                                            compare_op_t,
                                            policy_t>;
#else
  using dispatch_t = 
    hipcub::DispatchMergeSort<key_input_it_t, value_input_it_t, key_it_t, value_it_t, offset_t, compare_op_t>;
#endif

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> buffer_1(elements);
  thrust::device_vector<T> buffer_2(elements);

  gen(seed_t{}, buffer_1);

  key_t *d_buffer_1 = thrust::raw_pointer_cast(buffer_1.data());
  key_t *d_buffer_2 = thrust::raw_pointer_cast(buffer_2.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(elements);

  // Allocate temporary storage:
  std::size_t temp_size{};
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_buffer_1,
                       nullptr,
                       d_buffer_2,
                       nullptr,
                       static_cast<offset_t>(elements),
                       compare_op_t{},
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         d_buffer_1,
                         nullptr,
                         d_buffer_2,
                         nullptr,
                         static_cast<offset_t>(elements),
                         compare_op_t{},
                         launch.get_stream());
  });
}

NVBENCH_BENCH_TYPES(merge_sort_keys, NVBENCH_TYPE_AXES(all_value_types, offset_types))
  .set_name("hipcub::DeviceMergeSort::SortKeys")
  .set_type_axes_names({"T", "OffsetT"})
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

