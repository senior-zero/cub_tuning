#pragma once

#include <cub/device/device_reduce.cuh>

#include <thrust/device_vector.h>
#include <thrust/fill.h>

#include <string>

#include <nvbench/nvbench.cuh>

// %PARAM% TUNE_BLOCK_THREADS bt 128:256
// %PARAM% TUNE_ITEMS_PER_THREAD ipt 16:20
// %PARAM% TUNE_ITEMS_PER_VEC_LOAD ipv 1:2:4

template <typename AccumT, typename OffsetT>
struct policy_hub_t
{
  struct policy_t : hipcub::ChainedPolicy<300, policy_t, policy_t>
  {
    static constexpr int threads_per_block  = TUNE_BLOCK_THREADS;
    static constexpr int items_per_thread   = TUNE_ITEMS_PER_THREAD;
    static constexpr int items_per_vec_load = TUNE_ITEMS_PER_VEC_LOAD;

    using ReducePolicy = hipcub::AgentReducePolicy<threads_per_block,
                                                items_per_thread,
                                                AccumT,
                                                items_per_vec_load,
                                                hipcub::BLOCK_REDUCE_WARP_REDUCTIONS,
                                                hipcub::LOAD_DEFAULT>;

    // SingleTilePolicy
    using SingleTilePolicy = ReducePolicy;

    // SegmentedReducePolicy
    using SegmentedReducePolicy = ReducePolicy;
  };

  using MaxPolicy = policy_t;
};

template <typename T>
void reduce(nvbench::state &state, nvbench::type_list<T>)
{
  using accum_t     = T;
  using input_it_t  = const T *;
  using output_it_t = T *;
  using offset_t    = std::int32_t;
  using output_t    = T;
  using init_t      = T;
  using op_t        = hipcub::Sum;
  using policy_t    = policy_hub_t<accum_t, offset_t>;
  using dispatch_t =
    hipcub::DispatchReduce<input_it_t, output_it_t, offset_t, op_t, init_t, accum_t, policy_t>;

  // Retrieve axis parameters
  const auto elements = static_cast<std::size_t>(state.get_int64("Elements"));
  thrust::device_vector<T> in(elements);
  thrust::fill(in.begin(), in.begin() + elements / 2, T{1});
  thrust::device_vector<T> out(1);

  input_it_t d_in   = thrust::raw_pointer_cast(in.data());
  output_it_t d_out = thrust::raw_pointer_cast(out.data());

  // Enable throughput calculations and add "Size" column to results.
  state.add_element_count(elements);
  state.add_global_memory_reads<T>(elements, "Size");
  state.add_global_memory_writes<T>(1);

  // Allocate temporary storage:
  std::size_t temp_size;
  dispatch_t::Dispatch(nullptr,
                       temp_size,
                       d_in,
                       d_out,
                       static_cast<offset_t>(elements),
                       op_t{},
                       init_t{},
                       0 /* stream */);

  thrust::device_vector<nvbench::uint8_t> temp(temp_size);
  auto *temp_storage = thrust::raw_pointer_cast(temp.data());

  state.exec([&](nvbench::launch &launch) {
    dispatch_t::Dispatch(temp_storage,
                         temp_size,
                         d_in,
                         d_out,
                         static_cast<offset_t>(elements),
                         op_t{},
                         init_t{},
                         launch.get_stream());
  });
}

using all_value_types =
  nvbench::type_list<nvbench::int8_t, nvbench::int16_t, nvbench::int32_t, nvbench::int64_t, __int128>;

NVBENCH_BENCH_TYPES(reduce, NVBENCH_TYPE_AXES(all_value_types))
  .set_name("hipcub::DeviceReduce::Reduce")
  .add_int64_power_of_two_axis("Elements", nvbench::range(16, 28, 4));

